#include <stdio.h>

// Note: Needs compute capability > 2.0, so compile with:
// nvcc hello_world_02.cu -arch=compute_20 -code=sm_20,compute_20 -o hello_world_02.out
// Other notes: can have trouble when N is large...
// Default buffer is ~8MB
// cuCtxSetLimit(CU_LIMIT_PRINTF_FIFO_SIZE, ...)


#include <hip/hip_runtime.h>


#define N 20000
#define GRID_D1 20
#define GRID_D2 2
#define BLOCK_D1 512
#define BLOCK_D2 1
#define BLOCK_D3 1

__global__ void hello(void)
{
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    if (idx < N){  
        printf("Hello world! My block index is (%d,%d) [Grid dims=(%d,%d)], 3D-thread index within block=(%d,%d,%d) => thread index=%d\n", blockIdx.x, blockIdx.y, gridDim.x, gridDim.y, threadIdx.x, threadIdx.y, threadIdx.y, idx);
    } else {
        printf("Hello world! My block index is (%d,%d) [Grid dims=(%d,%d)], 3D-thread index within block=(%d,%d,%d) => thread index=%d [### this thread would not be used for N=%d ###]\n", blockIdx.x, blockIdx.y, gridDim.x, gridDim.y, threadIdx.x, threadIdx.y, threadIdx.y, idx, N);
    }
}


int main(int argc,char **argv)
{
    // Increase buffer size for illustration...
    size_t factor_increase = 4;
    // First, find buffer size:
    size_t buffer_size; 
    hipError_t stat;
    stat = hipDeviceGetLimit(&buffer_size,hipLimitPrintfFifoSize);
     printf("Buffer size = %u\n",buffer_size);	
    if (stat == hipSuccess){
        // Increase by a factor:
        stat = hipDeviceSetLimit(hipLimitPrintfFifoSize, buffer_size*factor_increase);
    	stat = hipDeviceGetLimit(&buffer_size,hipLimitPrintfFifoSize);
	printf("*NEW* Buffer size = %u\n",buffer_size);	
        if (stat == hipSuccess){
            printf("Successfully increased printf buffer size...\n");
        } else {
            printf("Failed to increase printf buffer size...\n");
        }
    } else {
        printf("Failed to retrieve printf buffer size...\n");
    }

    const dim3 blockSize(BLOCK_D1, BLOCK_D2, BLOCK_D3);
    const dim3 gridSize(GRID_D1, GRID_D2, 1);
    int nthreads = BLOCK_D1*BLOCK_D2*BLOCK_D3*GRID_D1*GRID_D2;
    if (nthreads < N){
        printf("\n============ NOT ENOUGH THREADS TO COVER N=%d ===============\n\n",N);
    } else {
        printf("Launching %d threads (N=%d)\n",nthreads,N);
    }
    
    // launch the kernel
    hello<<<gridSize, blockSize>>>();
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr){
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    } else {
        printf("kernel launch success!\n");
    }
    
    printf("That's all!\n");

    return 0;
}




