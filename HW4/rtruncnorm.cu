#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

extern "C"
{

__global__ void 
rtruncnorm_kernel(float *vals, int n, 
                  float *mu, float *sigma, 
                  float *lo, float *hi,
                  int rng_a, int rng_b,	int rng_c)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

    // Setup the RNG:
	hiprandState rng;
    hiprand_init(rng_a+idx*rng_b, rng_c, 0, &rng);
    
    // Sample:
    if (idx < n) {
	if (isfinite(lo[idx]) && isfinite(hi[idx])) {
		
		// sample from both finite
		float mu_neg = (lo[idx] - mu[idx])/sigma[idx];
		float mu_pos = (hi[idx] - mu[idx])/sigma[idx];
		float z = mu_neg + hiprand_uniform(&rng)*(mu_pos-mu_neg);
		float psi_z = expf(-z*z/2);
		if (mu_neg > 0) float psi_z = expf(-(mu_neg*mu_neg - z*z)/2);
		if (mu_pos < 0) float psi_z = expf(-(mu_pos*mu_pos - z*z)/2);
		float u = hiprand_uniform(&rng);
		while (u >= psi_z) {
			z = mu_neg + hiprand_uniform(&rng)*(mu_pos-mu_neg);
			psi_z = expf(-z*z/2);
			if (mu_neg > 0) psi_z = expf(-(mu_neg*mu_neg - z*z)/2);
			if (mu_pos < 0) psi_z = expf(-(mu_pos*mu_pos - z*z)/2);
			u = hiprand_uniform(&rng);
		}
		vals[idx] = sigma[idx]*z+mu[idx];
	}
	if (!isfinite(lo[idx])) {
		// sample from truncated norm -b to infinity, then reverse sign
		float mu_neg = (-hi[idx] - mu[idx])/sigma[idx];
		float alpha = (mu_neg + sqrtf(mu_neg*mu_neg + 4))/2;
		float expo_rand = logf(1 - hiprand_uniform(&rng))/(-alpha);
		float z = mu_neg + expo_rand;
		float psi_z = expf(-(mu_neg - alpha)*(mu_neg - alpha)/2 - (alpha - z)*(alpha - z)/2);
		if (mu_neg < alpha) float psi_z = expf(-(alpha - z)*(alpha - z)/2);
		float u = hiprand_uniform(&rng);
		while (u >= psi_z) {
			expo_rand = logf(1 - hiprand_uniform(&rng))/(-alpha);
			z = mu_neg + expo_rand;
			psi_z = expf(-(mu_neg - alpha)*(mu_neg - alpha)/2 - (alpha - z)*(alpha - z)/2);
			if (mu_neg < alpha) psi_z = expf(-(alpha - z)*(alpha - z)/2);
			u = hiprand_uniform(&rng);
		}
		vals[idx] = -(sigma[idx]*z+mu[idx]);
	}
	if (!isfinite(hi[idx])) {
		// sample from truncated norm a to infinity
		float mu_neg = (lo[idx] - mu[idx])/sigma[idx];
		float alpha = (mu_neg + sqrtf(mu_neg*mu_neg + 4))/2;
		float expo_rand = logf(1 - hiprand_uniform(&rng))/(-alpha);
		float z = mu_neg + expo_rand;
		float psi_z = expf(-(mu_neg - alpha)*(mu_neg - alpha)/2 - (alpha - z)*(alpha - z)/2);
		if (mu_neg < alpha) float psi_z = expf(-(alpha - z)*(alpha - z)/2);
		float u = hiprand_uniform(&rng);
		while (u >= psi_z) {
			expo_rand = logf(1 - hiprand_uniform(&rng))/(-alpha);
			z = mu_neg + expo_rand;
			psi_z = expf(-(mu_neg - alpha)*(mu_neg - alpha)/2 - (alpha - z)*(alpha - z)/2);
			if (mu_neg < alpha) psi_z = expf(-(alpha - z)*(alpha - z)/2);
			u = hiprand_uniform(&rng);
		}
		vals[idx] = sigma[idx]*z+mu[idx];
	}
	}
	return;
}

} // END extern "C"

